#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void add_ten_kernel(const float* vec, float* result, int maxlen) {
    int local_idx = threadIdx.x;
    if (local_idx < maxlen) {
        result[local_idx] = vec[local_idx] + 10;
    }
}

torch::Tensor add_ten(torch::Tensor vec) {
    const auto maxlen = vec.size(0);

    auto result = torch::empty_like(vec);

    add_ten_kernel<<<1, maxlen>>>(vec.data_ptr<float>(), result.data_ptr<float>(), maxlen);

    return result;
}
