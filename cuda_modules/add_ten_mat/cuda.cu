#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
__global__ void add_ten_mat_kernel(const float* input, float* result, int height, int width){

    // i know it's not necessary but i will try to use blocks here



    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    int col_idx = blockIdx.y * blockDim.y + threadIdx.y;



    if (row_idx < height && col_idx < width){

        int ndx = row_idx * width + col_idx;

        result[ndx] = input[ndx] + 10; 

    }







}



torch::Tensor add_ten_mat(torch::Tensor input){

    int height = input.size(0);

    int width = input.size(1);



    auto result = torch::empty_like(input);

    dim3 threads_per_block(16,16);

    

    dim3 num_blocks((width + threads_per_block.x - 1)/threads_per_block.x, (height + threads_per_block.y - 1)/threads_per_block.y);



    add_ten_mat_kernel<<<num_blocks, threads_per_block>>>(input.data_ptr<float>(), result.data_ptr<float>(), height, width);



    return result;



}