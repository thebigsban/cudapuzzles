#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 256
__global__ void vec_dot_prod_kernel(const float* a, const float* b, float* result, int len){
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gidx < len){
        result[gidx] = a[gidx] * b[gidx];
    }

    __syncthreads();
    
    if (threadIdx.x == 0){
        for (int i = 1; i < len; i++){
            result[0] += result[i];
        }
    }
}

torch::Tensor vec_dot_prod(torch::Tensor a, torch::Tensor b){
    int len = a.size(0);
    auto result = torch::empty_like(a);
    int num_blocks = (len + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
    vec_dot_prod_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(a.data_ptr<float>(),b.data_ptr<float>(), result.data_ptr<float>(), len);
    
    return result;
}