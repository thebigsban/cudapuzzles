#include "hip/hip_runtime.h"
__global__ void vec_sum_kernel(const float* a, const float* b, float* result, int len) {
    int local_idx = threadIdx.x;

    if (local_idx < len){
        result[local_idx] = a[local_idx] + b[local_idx];
    }
}

torch::Tensor vec_sum(torch::Tensor a, torch::Tensor b){
    int len = a.size(0);
    auto result = torch::empty_like(a);
    vec_sum_kernel<<<1, len>>>(a.data_ptr<float>(), b.data_ptr<float>(), result.data_ptr<float>(), len);
    return result;
}