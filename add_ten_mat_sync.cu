#include "hip/hip_runtime.h"
#define BLOCKSIZE 16

__global__ void add_ten_mat_sync_kernel(const float* input, float* result, int height, int width){
    // implemented with shared memory and thread synchronization as an exercise

    __shared__ float temp [BLOCKSIZE][BLOCKSIZE];


    int grow_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int gcol_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int lrow_idx = threadIdx.x;
    int lcol_idx = threadIdx.y;

    int ndx = grow_idx * width + gcol_idx;
    if (grow_idx < height && gcol_idx < width){
        temp[lrow_idx][lcol_idx] = input[ndx];
        // synchronize
        __syncthreads();   
    }

    if (grow_idx < height && gcol_idx < width){
        result[ndx] = temp[lrow_idx][lcol_idx] + 10;
    }
}

torch::Tensor add_ten_mat_sync(torch::Tensor input){

    int height = input.size(0);
    int width = input.size(1);

    auto result = torch::empty_like(input);
    dim3 threads_per_block(BLOCKSIZE,BLOCKSIZE);
    
    dim3 num_blocks((width + threads_per_block.x - 1)/threads_per_block.x, (height + threads_per_block.y - 1)/threads_per_block.y);

    add_ten_mat_sync_kernel<<<num_blocks, threads_per_block>>>(input.data_ptr<float>(), result.data_ptr<float>(), height, width);

    return result;

}