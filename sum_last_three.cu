#include "hip/hip_runtime.h"
#define SUMOVER 3
#define THREADS_PER_BLOCK 256
void __global__ sum_last_three_kernel(const float* input, float* result) {

    // initialize temporary shared array with all zeroes
    __shared__ float temp[THREADS_PER_BLOCK + SUMOVER - 1];

    // calculate global and local indices
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x;

    // the index of the temp array is longer than the number of outputs by SUMOVER-1
    int tindex = lindex + SUMOVER - 1;
    
    // read input into shared memory
    temp[tindex] = input[gindex];
    
    // synchronize threads
    __syncthreads();

    // number to keep track of sum
    float out = 0;

    for (int i = 0; i <= SUMOVER-1; i++){
        out += temp[lindex + i];
    }
    result[gindex] = out;
}


torch::Tensor sum_last_three(torch::Tensor input){

    int totallen = input.size(0);
    //int threads_per_block = 256;
    dim3 numblocks((totallen + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK);

    auto result = torch::empty_like(input);

    sum_last_three_kernel<<<numblocks, THREADS_PER_BLOCK>>>(input.data_ptr<float>(), result.data_ptr<float>());

    return result;

}