#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void add_ten_kernel(const float* input, float* result, const int* maxlen) {
    local_idx = threadIdx.x;
    if (local_idx < maxlen){
        result[local_idx] = input[local_idx] + 10;
    }
}

torch::Tensor add_ten(torch::Tensor vec){
    const auto maxlen = vec.size();

    auto result = torch::empty_like(vec);

    add_ten_kernel<<<1, maxlen>>>(vec.data_ptr<float>(), result.data_ptr<float>(), maxlen);

    return result;
}
